#include "hip/hip_runtime.h"
#define API_EXPORT


#include "gpu_kernel.h"
#include "matrix.h"
#include "helpers.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>

namespace
{
	const size_t BLOCK_SIZE = 32;
}

typedef struct Matrix
{
	size_t width;
	size_t height;
	size_t stride;
	float* elements;
};

__device__ float GetElement(const Matrix A, size_t row, size_t col)
{
	return A.elements[row * A.stride + col];
}

__device__ void SetElement(const Matrix A, size_t row, size_t col, float value)
{
	A.elements[row * A.stride + col] = value;
}

__device__ Matrix GetSubMatrix(Matrix A, size_t row, size_t col)
{
	Matrix Asub;
	Asub.width = BLOCK_SIZE;
	Asub.height = BLOCK_SIZE;
	Asub.stride = A.stride;
	Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
	return Asub;
}

__global__ void GPU_test_helper(const Matrix A, const Matrix B, Matrix C)
{
	size_t blockRow = blockIdx.y;
	size_t blockCol = blockIdx.x;

	Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

	float Cvalue = 0;

	size_t row = threadIdx.y;
	size_t col = threadIdx.x;

	for (size_t m = 0; m < (A.width / BLOCK_SIZE); ++m)
	{
		Matrix Asub = GetSubMatrix(A, blockRow, m);
		Matrix Bsub = GetSubMatrix(B, m, blockCol);

		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

		As[row][col] = GetElement(Asub, row, col);
		Bs[row][col] = GetElement(Bsub, row, col);
		__syncthreads();

		for (size_t e = 0; e < BLOCK_SIZE; ++e)
		{
			Cvalue += As[row][e] * Bs[e][col];
		}
		__syncthreads();
	}
	SetElement(Csub, row, col, Cvalue);

	/*int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	C.elements[row*C.width + col] = A.elements[row * A.width + col] + B.elements[row * B.width + col];
	*/
}

__host__ void WhatAnError(hipError_t er)
{
	switch (er)
	{
		//hipSuccess
	case 0: std::cerr << "CUDA SUCCESS." << std::endl; break;
		//hipErrorMissingConfiguration
	case 1: std::cerr << "CUDA ERROR MISSING CONFIGURATION." << std::endl; break;
		//hipErrorOutOfMemory
	case 2: std::cerr << "CUDA ERROR MEMORY ALLOCATION." << std::endl; break;
		//hipErrorNotInitialized
	case 3: std::cerr << "CUDA ERROR INITIALIZATION ERROR." << std::endl; break;
		//hipErrorLaunchFailure
	case 4: std::cerr << "CUDA ERROR LAUNCH FAILURE." << std::endl; break;
		//hipErrorPriorLaunchFailure
	case 5: std::cerr << "CUDA ERROR PRIOR LAUNCH FAILURE." << std::endl; break;
		//hipErrorLaunchTimeOut
	case 6: std::cerr << "CUDA ERROR LAUNCH TIMEOUT." << std::endl; break;
		//unknowError
	default: std::cerr << "CUDA ERROR CODE: " << er << std::endl; break;
	}
}

__global__ void DisplayMatrixKernel(Matrix const& Mat)
{
	for (int i = 0; i < Mat.height; i++)
	{
		for (int j = 0; j < Mat.width; j++)
		{
			printf("%f", Mat.elements[i * Mat.width + j]);
		}
		printf("\n");
	}
}

GPU_API_EXPORT  void DisplayMatrix(double const* matrix, size_t size, char* name)
{
	std::cout << std::endl << name << std::endl;
	for (int i = 0; i < size*size; i++)
	{
		if (i % size == 0)
		{
			std::cout << std::endl;
		}
		std::cout << matrix[i] << " ";
	}

	std::cout << std::endl;
}

GPU_API_EXPORT  void GPU_test(PerfComparison::Matrix<float> const& A, PerfComparison::Matrix<float> const& B, PerfComparison::Matrix<float> &C, float& ComputingTime)
{
	hipError_t error;
	hipEvent_t start, stop;
	Matrix device_A, device_B, device_C;
	size_t bytesNumber;

	device_A.height = A.rows();
	device_A.width = A.columns();
	device_A.stride = A.columns();
	bytesNumber = device_A.width * device_A.height * sizeof(float);
	error = hipMalloc(&device_A.elements, bytesNumber);
	WhatAnError(error);
	error = hipMemcpy(device_A.elements, A.get(), bytesNumber, hipMemcpyHostToDevice);
	WhatAnError(error);


	device_B.height = B.rows();
	device_B.width = B.columns();
	device_B.stride = B.columns();
	bytesNumber = device_B.width * device_B.height * sizeof(float);
	error = hipMalloc(&device_B.elements, bytesNumber);
	WhatAnError(error);
	error = hipMemcpy(device_B.elements, B.get(), bytesNumber, hipMemcpyHostToDevice);
	WhatAnError(error);

	device_C.height = C.rows();
	device_C.width = C.columns();
	device_C.stride = C.columns();
	bytesNumber = device_C.width * device_C.height * sizeof(float);
	error = hipMalloc(&device_C.elements, bytesNumber);
	WhatAnError(error);

	error = hipEventCreate(&start);
	WhatAnError(error);
	error = hipEventCreate(&stop);
	WhatAnError(error);

	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(device_B.width / threads.x, device_A.height / threads.y);

	error = hipEventRecord(start, 0);
	WhatAnError(error);

	/*
	std::cout << "matrix A" << std::endl;
	DisplayMatrixKernel<<<1,1>>>(device_A);
	std::cout << "matrix B" << std::endl;
	DisplayMatrixKernel<<<1,1>>>(device_B);
	*/

	GPU_test_helper <<<blocks, threads >>>(device_A, device_B, device_C);


	error = hipDeviceSynchronize();
	//error = hipDeviceSynchronize();
	WhatAnError(error);
	error = hipEventRecord(stop, 0);
	WhatAnError(error);
	error = hipEventSynchronize(stop);
	WhatAnError(error);

	error = hipEventElapsedTime(&ComputingTime, start, stop);
	WhatAnError(error);


	error = hipMemcpy(C.get(), device_C.elements, bytesNumber, hipMemcpyDeviceToHost);
	WhatAnError(error);


	error = hipFree(device_A.elements);
	WhatAnError(error);
	error = hipFree(device_B.elements);
	WhatAnError(error);
	error = hipFree(device_C.elements);
	WhatAnError(error);
	error = hipEventDestroy(start);
	WhatAnError(error);
	error = hipEventDestroy(stop);
	WhatAnError(error);
}